#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
// #include <ATen/cuda/detail/PhiloxCudaStateRaw.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <vector>

#define MAX_GROUP_SIZE 32

#define PRUNE_MAX 0
#define PRUNE_RND 1

//#define UINT_MAX (__INT_MAX__ * 2U + 1)
#define UNROLL 4
#define idx_4d(b, c, w, h, C, W, H)                                            \
  ((b) * (C) * (H) * (W) + (c) * (H) * (W) + (w) * (H) + (h))
#define get_0dim(idx, C, W, H) (idx / ((C) * (W) * (H)))
#define get_1dim(idx, C, W, H) ((idx / (W) / (H)) % (C))
#define get_2dim(idx, C, W, H) ((idx / (H)) % (W))
#define get_3dim(idx, C, W, H) (idx % (H))

namespace {


template <typename scalar_t>
__device__ void abs_bubble_sort(const scalar_t *__restrict__ vals,
                                int32_t *idxs, const int32_t size) {
  for (int32_t i = 0; i < size; i++) {
    for (int32_t j = 0; j < size - i - 1; j++) {
      if (abs(vals[idxs[j]]) < abs(vals[idxs[j + 1]])) {
        int32_t tmp = idxs[j];
        idxs[j] = idxs[j + 1];
        idxs[j + 1] = tmp;
      }
    }
  }
}

// prune_max_channelwise
template <typename scalar_t>
__device__ void
prune_max_channelwise(const scalar_t *__restrict__ x, scalar_t *__restrict__ y,
                      const int32_t nonzero, const int32_t group_size,
                      const int32_t B, const int32_t C, const int32_t W,
                      const int32_t H) {
  const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t b = get_0dim(idx, C, W, H);
  const int32_t c = get_1dim(idx, C, W, H);
  const int32_t w = get_2dim(idx, C, W, H);
  const int32_t h = get_3dim(idx, C, W, H);

  if (c >= C / group_size) {
    return;
  }

  int32_t heap[MAX_GROUP_SIZE];

  for (int32_t i = 0; i < group_size; i++) {
    int32_t ci = c * group_size + i;
    heap[i] = idx_4d(b, ci, w, h, C, W, H);
  }

  abs_bubble_sort<scalar_t>(x, heap, group_size);

  // assign nonzero elements to y
  for (int32_t i = 0; i < nonzero; i++) {
    y[heap[i]] = x[heap[i]];
  }
}

// prune_max_batchwise
template <typename scalar_t>
__device__ void
prune_max_batchwise(const scalar_t *__restrict__ x, scalar_t *__restrict__ y,
                    const int32_t nonzero, const int32_t group_size,
                    const int32_t B, const int32_t C, const int32_t W,
                    const int32_t H) {
  const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t b = get_0dim(idx, C, W, H);
  const int32_t c = get_1dim(idx, C, W, H);
  const int32_t w = get_2dim(idx, C, W, H);
  const int32_t h = get_3dim(idx, C, W, H);

  if (b >= B / group_size) {
    return;
  }

  int32_t heap[MAX_GROUP_SIZE];

  for (int32_t i = 0; i < group_size; i++) {
    int32_t bi = b * group_size + i;
    heap[i] = idx_4d(bi, c, w, h, C, W, H);
  }

  abs_bubble_sort<scalar_t>(x, heap, group_size);

  // assign nonzero elements to y
  for (int32_t i = 0; i < nonzero; i++) {
    y[heap[i]] = x[heap[i]];
  }
}

template <typename scalar_t>
__device__ void
prune_rnd_batchwise(const scalar_t *__restrict__ x, scalar_t *__restrict__ y,
                    hiprandStatePhilox4_32_10_t *state,
                    const int32_t nonzero, const int32_t group_size,
                    const int32_t B, const int32_t C, const int32_t W,
                    const int32_t H) {
  const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t b = get_0dim(idx, C, W, H);
  const int32_t c = get_1dim(idx, C, W, H);
  const int32_t w = get_2dim(idx, C, W, H);
  const int32_t h = get_3dim(idx, C, W, H);

  if (b >= B / group_size) {
    return;
  }

  int32_t heap[MAX_GROUP_SIZE];

  for (int32_t i = 0; i < group_size; i++) {
    int32_t bi = b * group_size + i;
    heap[i] = idx_4d(bi, c, w, h, C, W, H);
  }

  // Randomly shuffle the heap
  for (int32_t i = 0; i < group_size / UNROLL; i++) {
    float4 rand = hiprand_uniform4(state);
    for (int32_t j = 0; j < UNROLL; j++) {
      int pos = i * UNROLL + j;
      int32_t r = (int32_t)((&rand.x)[j] * group_size);
      
      int32_t tmp = heap[pos];
      heap[pos] = heap[r];
      heap[r] = tmp;
    }
  }

  // assign nonzero elements to y
  for (int32_t i = 0; i < nonzero; i++) {
    y[heap[i]] = x[heap[i]];
  }
}

template <typename scalar_t>
__device__ void
prune_rnd_channelwise(const scalar_t *__restrict__ x, scalar_t *__restrict__ y,
                      hiprandStatePhilox4_32_10_t *state,
                      const int32_t nonzero, const int32_t group_size,
                      const int32_t B, const int32_t C, const int32_t W,
                      const int32_t H) {
  const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t b = get_0dim(idx, C, W, H);
  const int32_t c = get_1dim(idx, C, W, H);
  const int32_t w = get_2dim(idx, C, W, H);
  const int32_t h = get_3dim(idx, C, W, H);

  if (b >= B || c >= C / group_size || w >= W || h >= H) {
    return;
  }

  int32_t heap[MAX_GROUP_SIZE];

  // Randomly select nonzero elements
  for (int32_t i = 0; i < group_size; i++) {
    int32_t ci = c * group_size + i;
    // printf("idx(%d) %d\n", idx, ci);
    heap[i] = idx_4d(b, ci, w, h, C, W, H);
  }

  // Randomly shuffle the heap
  for (int32_t i = 0; i < group_size / UNROLL; i++) {
    float4 rand = hiprand_uniform4(state);
    for (int32_t j = 0; j < UNROLL; j++) {
      int pos = i * UNROLL + j;
      int32_t r = (int32_t)((&rand.x)[j] * group_size);
      
      int32_t tmp = heap[pos];
      heap[pos] = heap[r];
      heap[r] = tmp;
    }
  }

  // assign nonzero elements to y
  for (int32_t i = 0; i < nonzero; i++) {
    y[heap[i]] = x[heap[i]];
  }
}

template <typename scalar_t>
__global__ void
prune_kernel(const scalar_t *__restrict__ x, scalar_t *__restrict__ y,
             at::PhiloxCudaState philox_args,
             const int32_t prune_type, const int32_t prune_dim,
             const int32_t nonzero, const int32_t group_size, const int32_t B,
             const int32_t C, const int32_t W, const int32_t H) {
  const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t c = get_1dim(idx, C, W, H);

  // if (idx >= B * W * H * C) {
  //   return;
  // }
  auto seeds = at::cuda::philox::unpack(philox_args);
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds), idx, std::get<1>(seeds), &state);

  // Select proper pruning algorithm and call it
  if (prune_type == PRUNE_MAX && prune_dim == 0) {
    prune_max_batchwise<scalar_t>(x, y, nonzero, group_size, B, C, W, H);
  } else if (prune_type == PRUNE_MAX && prune_dim == 1) {
    prune_max_channelwise<scalar_t>(x, y, nonzero, group_size, B, C, W, H);
  } else if (prune_type == PRUNE_RND && prune_dim == 0) {
    prune_rnd_batchwise<scalar_t>(x, y, &state, nonzero, group_size, B, C, W, H);
  } else if (prune_type == PRUNE_RND && prune_dim == 1) {
    prune_rnd_channelwise<scalar_t>(x, y, &state, nonzero, group_size, B, C, W, H);
  }
}
} // namespace


at::Tensor prune_cuda(const at::Tensor x, const int prune_type,
                      const int prune_dim, const int nonzero,
                      const int group_size) {
  const auto B = x.size(0);
  const auto C = x.size(1);
  const auto W = x.size(2);
  const auto H = x.size(3);
  const auto size = B * C * W * H;
  auto y = at::zeros_like(x);
  const int threads = 256;
  int blocks;
  if (prune_dim == 0) {
    blocks = ((B / group_size) * C * W * H) / threads + 1;
  } else if (prune_dim == 1) {
    blocks = ((C / group_size) * B * W * H) / threads + 1;
  } else {
    AT_ERROR("prune_dim must be 0 or 1");
  }

  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator(x.get_device()));
  int32_t counter_offset = 1;
  at::PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }

  AT_DISPATCH_FLOATING_TYPES(x.type(), "prune_cuda", ([&] {
                               prune_kernel<scalar_t><<<blocks, threads>>>(
                                   x.data<scalar_t>(), y.data<scalar_t>(), 
                                   rng_engine_inputs,
                                   prune_type, prune_dim, nonzero, group_size,
                                   B, C, W, H);
                             }));
                         
  return y;
}
